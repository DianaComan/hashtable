#include "hip/hip_runtime.h"
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <ctime>
#include <sstream>
#include <string>
#include "gpu_hashtable.hpp"

/*functie hash*/
__device__ long long getHash(int x, int N) {
        if (x < 0) x = -x;
        x = ((x >> 16) ^ x) * HASH_NO;
    	x = ((x >> 16) ^ x) * HASH_NO;
    	x = (x >> 16) ^ x;
		x = x % N;
    	return x;
}

/*copie o valoare a unei chei dintr-un hashtable in altul*/
__global__ void reshape_hashT(hashTable h1, long siz1, hashTable h2, long siz2) {
	int vall, idx = blockIdx.x * blockDim.x + threadIdx.x, key1, key2;
	bool ok = false;
	if ((h1.pairs[idx].key == KEY_INVALID) || (siz1 <= idx))
		return;
	key2 = h1.pairs[idx].key;
	vall = getHash(key2, h2.size);
	for (int i = vall; i < siz2; i++) {
		key1 = atomicCAS(&h2.pairs[i].key, KEY_INVALID, key2);
		if (key1 == KEY_INVALID) {
			h2.pairs[i].value = h1.pairs[idx].value;
			ok = true;
			break;
		} 
	}
	if (!ok) {
		for (int i = 0; i <vall; i++) {
			key1 = atomicCAS(&h2.pairs[i].key, KEY_INVALID, key2);
			if (key1 == KEY_INVALID) {
				h2.pairs[i].value = h1.pairs[idx].value;
				break;
			}

		}
	}
}

/*se insereaza o pereche de (key, value) in hashtable, daca nu se gaseste pe pozitia
 *returnata de functia hash, se va cauta pana la maximul posibil un loc liber, apoi
 *se cauta de la 0 la valoarea functiei
 */
__global__ void insert_hash(int k, int *keys, int *values, hashTable h, long siz) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x, key, vall;
	
	if (k <= idx) return;
	vall = getHash(keys[idx], siz);
	for (int i = vall; i < siz; i++) {
		key = atomicCAS(&h.pairs[i].key, KEY_INVALID, keys[idx]);
		if (key == KEY_INVALID || key == keys[idx]) {
			h.pairs[i].value = values[idx];
			return;
		} 
	}
	for (int i = 0; i < vall; i++) {
		key = atomicCAS(&h.pairs[i].key, KEY_INVALID, keys[idx]);
		if (key == KEY_INVALID || key == keys[idx]) {
			h.pairs[i].value = values[idx];
			return;
		}
	}
}

/*cauta valoarea unei chei in hashtable: se apeleaza functia hash, si daca 
 *valoarea nu se gaseste acolo, se face asemenator cu insert: se cauta pana
 *la dimensiunea maxima, apoi de la 0 la valoarea de hash
 */
__global__ void get_hash(int k, int *keys, int *values, hashTable h, long siz) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x, vall;
	if (k<=idx) return;
	vall = getHash(keys[idx], siz);
	for (int i = vall; i < siz; i++) {
		if (h.pairs[i].key == keys[idx]) {
			values[idx] = h.pairs[i].value;
			return;
		} 
	}
	for (int i = 0; i < vall; i++) {
		if (h.pairs[i].key == keys[idx]) {
			values[idx] = h.pairs[i].value;
			return;
		}
	}
}

/* INIT HASH
 */
GpuHashTable::GpuHashTable(int size) {
	hashT.size = size;
	cntPairs = 0;
	hashT.pairs = nullptr;
	hipMalloc(&hashT.pairs, size * sizeof(pair));
	hipMemset(hashT.pairs, 0, size * sizeof(pair));
}

/* DESTROY HASH
 */
GpuHashTable::~GpuHashTable() {
	hipFree(hashT.pairs);
}

/* RESHAPE HASH
 * cresc marimea noului table si se apeleaza reshape_hashT pt a muta valorile
 */
void GpuHashTable::reshape(int numBucketsReshape) {
	int k = hashT.size / THREADS_NO;  
	if (!(hashT.size % THREADS_NO == 0)) k = k + 1;
	hashTable newH;
	newH.size = numBucketsReshape;

	hipMalloc(&newH.pairs, numBucketsReshape * sizeof(pair));
	hipMemset(newH.pairs, 0, numBucketsReshape * sizeof(pair));
	reshape_hashT<<< k, THREADS_NO >>>(hashT, hashT.size, newH, newH.size);

	hipDeviceSynchronize();
	hipFree(hashT.pairs);
	hashT = newH;
}

/* INSERT BATCH
 * se insereaza perechile, dandu-se reshape la hashtable daca e necesar
 * prin hipMemcpy se pun datele in VRAM
 */
bool GpuHashTable::insertBatch(int *keys, int *values, int numKeys) {
	int *aKeys, *aValues, k = numKeys / THREADS_NO, nr = cntPairs + numKeys;
	if (numKeys % THREADS_NO != 0) k++;   
	hipMalloc(&aKeys, numKeys * sizeof(int));
	hipMalloc(&aValues, numKeys * sizeof(int));

	if (nr / hashT.size >= LOADFACTOR_MAX) reshape((int) (nr / LOADFACTOR_MIN));

	hipMemcpy(aKeys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(aValues, values, numKeys * sizeof(int), hipMemcpyHostToDevice);
	insert_hash<<< k, THREADS_NO>>>(numKeys, aKeys, aValues, hashT, hashT.size);

	hipDeviceSynchronize();
	hipFree(aKeys);
	hipFree(aValues);
	cntPairs += numKeys;
	return true;
}

/* GET BATCH
 * se obtine valoarea apeland get_hash pt cheile din keys in valls
 * prin hipMemcpy se pun datele in VRAM
 */
int *GpuHashTable::getBatch(int *keys, int numKeys) {
	int *aKeys, *valls, k = numKeys / THREADS_NO;
	if (!(numKeys % THREADS_NO == 0)) k++;
	hipMalloc(&aKeys, numKeys * sizeof(int));
	hipMallocManaged(&valls, numKeys * sizeof(int));

	hipMemcpy(aKeys, keys, numKeys * sizeof(int), hipMemcpyHostToDevice);
	get_hash<<<k, THREADS_NO >>>(numKeys, aKeys, valls, hashT, hashT.size);
	hipDeviceSynchronize();
	hipFree(aKeys);
	return valls;
}

/* GET LOAD FACTOR
 * num elements / hash total slots elements
 */
float GpuHashTable::loadFactor() {
	if (hashT.size == 0) return 0;
	return (float(cntPairs) / hashT.size);
}

/*********************************************************/

#define HASH_INIT GpuHashTable GpuHashTable(1);
#define HASH_RESERVE(size) GpuHashTable.reshape(size);

#define HASH_BATCH_INSERT(keys, values, numKeys) GpuHashTable.insertBatch(keys, values, numKeys)
#define HASH_BATCH_GET(keys, numKeys) GpuHashTable.getBatch(keys, numKeys)

#define HASH_LOAD_FACTOR GpuHashTable.loadFactor()

#include "test_map.cpp"
